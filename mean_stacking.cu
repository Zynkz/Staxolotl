#include "hip/hip_runtime.h"
#include "mean_stacking.h"

void init_rand_matrix_ms(int* matrix, int width, int height){
    srand(0);
    for(int i = 0; i < width * height; i++){
        matrix[i] = rand()%10;
    }

}

void init_matrix_ms(int* matrix, int width, int height){
    for(int itr = 0; itr < width * height; itr++){
        matrix[itr] = 0;
    }
}

void mean_stack(int* result, int* next, int width, int height, int num_images){
    for(int itr = 0; itr < width * height; itr++){
        next[itr] = next[itr]/num_images;
    }
    for(int itr = 0; itr < width * height; itr++){
        result[itr] += next[itr];
    }
}

__global__ void initRandMatrixMs(int* matrix, int width, int height){
    int itr = threadIdx.x;
    matrix[itr] = clock()%10;
}

__global__ void initMatrixMs(int* matrix, int width, int height){
    int itr = threadIdx.x;
    matrix[itr] = 0;
}

__global__ void meanStack(int* result, int* next, int width, int height, int num_images){
    int itr = threadIdx.x;
    result[itr] += next[itr]/num_images;
}

int verified(){
    int array1[9] = {137, 432, 96, 12, 198, 323, 54, 2, 45};
    int array2[9] = {73, 109, 92, 129, 98, 363, 5, 23, 33};
    int result[9] = {104, 270, 94, 70, 148, 342, 29, 12, 38};
    int* test = (int*)malloc(sizeof(int)*9);

    int* d_array1;
    int* d_array2;
    int* d_result;

    hipMalloc((void**)&d_array1,  sizeof(int) * 9);
    hipMalloc((void**)&d_array2,  sizeof(int) * 9);
    hipMalloc((void**)&d_result,  sizeof(int) * 9);

    hipMemcpy(d_array1, array1, sizeof(int)*9, hipMemcpyHostToDevice);
    hipMemcpy(d_array2, array2, sizeof(int)*9, hipMemcpyHostToDevice);
    
    initRandMatrixMs<<<9,9>>>(d_result, 3, 3);
    hipMemcpy(test, d_result, sizeof(int)*9, hipMemcpyHostToDevice);
    for(int itr = 0; itr < 9; itr++){
        printf("%d ", test[itr]);
    }
    printf("\n");
    initMatrixMs<<<9,9>>>(d_result, 3, 3);

    hipMemcpy(test, d_result, sizeof(int)*9, hipMemcpyHostToDevice);
    for(int itr = 0; itr < 9; itr++){
        printf("%d ", test[itr]);
        if(test[itr] != 0){
            printf("\n");
            return 0;
        }
    }
    printf("\n");

    meanStack<<<9,9>>>(d_result, d_array1, 3, 3, 2);
    meanStack<<<9,9>>>(d_result, d_array2, 3, 3, 2);
    hipMemcpy(test, d_result, sizeof(int)*9, hipMemcpyHostToDevice);

    for(int itr = 0; itr < 9; itr++){
        printf("%d ", test[itr]);
        if(test[itr] != result[itr]){
            printf("\n");
            return 0;
        }
    }
    printf("\n");

    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(result);

    return 1;
}

double mean_stack_timed_test(){
    struct timespec time_start;
    struct timespec time_end;

    struct timespec total_time_start;
    struct timespec total_time_end;

    int num_images = 100;
    int width = 5496;
    int height = 3672;
    int N = width*height;

    if(!verified()){
        printf("Verification Unsucessful\n");
        exit(1);
    }else{
        printf("Verification Sucessful\n");
    }

    srand(0);

    clock_gettime(CLOCK_REALTIME, &total_time_start);

    int* d_result;
    hipMalloc((void**)&d_result,  sizeof(int) * width * height);
    initMatrixMs<<<N,N>>>(d_result, width, height);

    clock_gettime(CLOCK_REALTIME, &time_start);
    for(int itr = 0; itr < num_images; itr++){

        int* d_image;
        hipMalloc((void**)&d_image,  sizeof(int) * width * height);
        initRandMatrixMs<<<N,N>>>(d_image, width, height);

        meanStack<<<N, N>>>(d_result, d_image, width, height, num_images);
        hipFree(d_image);
    }
    clock_gettime(CLOCK_REALTIME, &time_end);

    hipFree(d_result);

    clock_gettime(CLOCK_REALTIME, &total_time_end);
    printf("\nCUDA time: %lfs\n", (time_end.tv_sec - time_start.tv_sec) + ((double)(time_end.tv_nsec - time_start.tv_nsec)/1000000000));
    printf("\nTotal time: %lfs\n", (total_time_end.tv_sec - total_time_start.tv_sec) + ((double)(total_time_end.tv_nsec - total_time_start.tv_nsec)/1000000000));
    return (time_end.tv_sec - time_start.tv_sec) + ((double)(time_end.tv_nsec - time_start.tv_nsec)/1000000000);
}

// int main(){
//     mean_stack_timed_test();
//     return 0;
// }